#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include "/usr/local/cuda/samples/common/inc/hip/hip_vector_types.h"

#define X_MIN (-1.5f)
#define X_MAX 1.5f
#define Y_MIN (-3.2f)
#define Y_MAX 2.0f

#define X_MIN_SAMPLE (-2.08f)
#define X_MAX_SAMPLE 1.08f
#define Y_MAX_SAMPLE 1.77f

#define X_DIM %(XDIM)s
#define Y_DIM %(YDIM)s
#define ITERS %(ITERS)s

__constant__ float X_SCALE = 1/(X_MAX - X_MIN) * X_DIM;
__constant__ float Y_SCALE = 1/(Y_MAX - Y_MIN) * Y_DIM;
__constant__ static float2 xy_min = (float2){X_MIN, Y_MIN};
__constant__ static float2 xy_scale = (float2){
	1/(X_MAX - X_MIN) * X_DIM,
	1/(Y_MAX - Y_MIN) * Y_DIM
};

//rotate/flip a quadrant appropriately
__device__
void rot(int n, int *x, int *y, int rx, int ry) {
    if (ry == 0) {
        if (rx == 1) {
            *x = n-1 - *x;
            *y = n-1 - *y;
        }

        //Swap x and y
        int t  = *x;
        *x = *y;
        *y = t;
    }
}

//convert (x,y) to d
__device__
int xy2d_hilbert (int n, int x, int y) {
    int rx, ry, s, d=0;
    for (s=n/2; s>0; s/=2) {
        rx = (x & s) > 0;
        ry = (y & s) > 0;
        d += s * s * ((3 * rx) ^ ry);
        rot(s, &x, &y, rx, ry);
    }
    return d;
}

//convert d to (x,y)
__device__
void d2xy_hilbert(int n, int d, int *x, int *y) {
    int rx, ry, s, t=d;
    *x = *y = 0;
    for (s=1; s<n; s*=2) {
        rx = 1 & (t/2);
        ry = 1 & (t ^ rx);
        rot(s, x, y, rx, ry);
        *x += s * rx;
        *y += s * ry;
        t /= 4;
    }
}

__device__
int xy2d (int xd, int yd, int x, int y) {
	int dim = 16;
	int block = x/dim*(yd/dim) + y/dim;
	int blockRow = x %% dim;
	int blockCol = y %% dim;
	return block*dim*dim + blockCol*dim + blockRow;
//	return block;
}


__device__ int to_pixel(float2 &temp, int2 &ixy) {
	temp -= xy_min;
	temp *= xy_scale;
	ixy = make_int2(temp);
	return xy2d(X_DIM, Y_DIM, ixy.x, ixy.y);
}

__device__
void write_pixel(float2 temp, int2 ixy,
	float4 z, unsigned int *canvas) {
	temp.x = z.y;
	temp.y = z.x;
	int idx = to_pixel(temp, ixy);
	if (0 <= ixy.x & ixy.x < X_DIM & 0 <= ixy.y & ixy.y < Y_DIM) {
	// if (0 <= idx & idx < X_DIM*Y_DIM) {
		// atomicAdd(&(canvas[ixy.y*X_DIM + ixy.x]), 1);
		atomicAdd(&(canvas[idx]), 1);
		// canvas[ixy.y*X_DIM + ixy.x] = idx;
		// atomicAdd(&(canvas[(ixy.y+1)*X_DIM - ixy.x-1]), 1);
	}
}

__device__
void generate_random_complex(float2 temp,
	float4 &z, float &dist, unsigned int &count) {

	temp.x *= X_MAX_SAMPLE-X_MIN_SAMPLE;
	temp.x += X_MIN_SAMPLE;
	temp.y *= Y_MAX_SAMPLE;

	z.x = temp.x;
	z.y = temp.y;
	z.z = temp.x;
	z.w = temp.y;
	dist = 0;
	count = 0;
}

__device__
bool check_bulbs(float4 z) {
	float zw2 = z.w*z.w;
	bool main_card = !(((z.z-0.25)*(z.z-0.25)
		+ (zw2))*(((z.z-0.25)*(z.z-0.25)
		+ (zw2))+(z.z-0.25)) < 0.25* zw2);
	bool period_2 = !((z.z+1.0) * (z.z+1.0) + (zw2) < 0.0625);
	bool smaller_bulb = !((z.z+1.309)*(z.z+1.309) + zw2 < 0.00345);
	bool smaller_bottom = !((z.z+0.125)*(z.z+0.125)
		+ (z.w-0.744)*(z.w-0.744) < 0.0088);
	bool smaller_top = !((z.z+0.125)*(z.z+0.125)
		+ (z.w+0.744)*(z.w+0.744) < 0.0088);
	return main_card & period_2 & smaller_bulb & smaller_bottom & smaller_top;
}

__device__ __forceinline__
void write_to_image(float4 z, float2 temp, int2 ixy, 
	int count, unsigned int *canvas) {
	z.x = z.z;
	z.y = z.w;
	for (int j = 0; j < count; j++) {
		temp.x = z.x*z.x - z.y*z.y + z.z;
		temp.y = 2*z.x*z.y + z.w;
		z.x = temp.x;
		z.y = temp.y;
		write_pixel(temp, ixy, z, canvas);
	}
}

extern "C" {
__global__
void buddha_kernel(unsigned int *canvas, int seed, float gridSize) {
	int idx = blockIdx.x 
		+ threadIdx.x * gridDim.x 
		+ threadIdx.y * gridDim.x * blockDim.x;

	int2 ixy;
	float2 temp, coord;
	unsigned int count;
	float4 z;
	float dist;
	hiprandState_t s;
	hiprand_init(seed, idx, 0, &s);

	for (coord.x = 0; coord.x < 1; coord.x += gridSize) {
		for (coord.y = 0; coord.y < 1; coord.y += gridSize) {

			for(int i = 0; i < 1; i++) {

				temp.x = hiprand_uniform(&s);
				temp.y = hiprand_uniform(&s);
				temp *= gridSize;
				temp += coord;

				generate_random_complex(temp, z, dist, count);
				if (check_bulbs(z)) {
					while (count < ITERS & dist < 4) {
						count++;
						temp.x = z.x*z.x - z.y*z.y + z.z;
						temp.y = 2*z.x*z.y + z.w;
						z.x = temp.x;
						z.y = temp.y;
						dist = z.x*z.x + z.y*z.y;
					}

					if (dist > 4) {
						write_to_image(z, temp, ixy, count, canvas);
						z.w *= -1;
						write_to_image(z, temp, ixy, count, canvas);						 
					}
				}
			}
			__syncthreads();
		}
	}
}
}