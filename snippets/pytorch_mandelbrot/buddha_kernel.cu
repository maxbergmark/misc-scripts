#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include "/usr/local/cuda/samples/common/inc/hip/hip_vector_types.h"

#define X_MIN (-1.5f)
#define X_MAX 1.5f
#define Y_MIN (-3.2f)
#define Y_MAX 2.0f

#define X_MIN_SAMPLE (-2.1f)
#define X_MAX_SAMPLE 1.1f
#define Y_MIN_SAMPLE 0.f
#define Y_MAX_SAMPLE 1.8f

#define X_DIM %(XDIM)s
#define Y_DIM %(YDIM)s
#define ITERS %(ITERS)s

__constant__ float X_SCALE = 1/(X_MAX - X_MIN) * X_DIM;
__constant__ float Y_SCALE = 1/(Y_MAX - Y_MIN) * Y_DIM;
__constant__ static float2 xy_min = (float2){X_MIN, Y_MIN};
__constant__ static float2 xy_scale = (float2){
	1/(X_MAX - X_MIN) * X_DIM,
	1/(Y_MAX - Y_MIN) * Y_DIM
};

__device__ void to_pixel(float2 &temp, int2 &ixy) {
	temp -= xy_min;
	temp *= xy_scale;
	ixy = make_int2(temp);
}

__device__
void write_pixel(float2 temp, int2 ixy,
	float4 z, unsigned int *canvas) {
	temp.x = z.y;
	temp.y = z.x;
	to_pixel(temp, ixy);
	if (0 <= ixy.x & ixy.x < X_DIM & 0 <= ixy.y & ixy.y < Y_DIM) {
		atomicAdd(&(canvas[ixy.y*X_DIM + ixy.x]), 1);
	}
}

__device__
void generate_random_complex(float2 temp,
	float4 &z, float &dist, unsigned int &count) {

	temp.x *= X_MAX_SAMPLE-X_MIN_SAMPLE;
	temp.x += X_MIN_SAMPLE;
	temp.y *= Y_MAX_SAMPLE-Y_MIN_SAMPLE;
	temp.y += Y_MIN_SAMPLE;

	z.x = temp.x;
	z.y = temp.y;
	z.z = temp.x;
	z.w = temp.y;
	dist = 0;
	count = 0;
}

__device__
bool check_bulbs(float4 z) {
	float zw2 = z.w*z.w;
	bool main_card = !(((z.z-0.25)*(z.z-0.25)
		+ (zw2))*(((z.z-0.25)*(z.z-0.25)
		+ (zw2))+(z.z-0.25)) < 0.25* zw2);
	bool period_2 = !((z.z+1.0) * (z.z+1.0) + (zw2) < 0.0625);
	bool smaller_bulb = !((z.z+1.309)*(z.z+1.309) + zw2 < 0.00345);
	bool smaller_bottom = !((z.z+0.125)*(z.z+0.125)
		+ (z.w-0.744)*(z.w-0.744) < 0.0088);
	bool smaller_top = !((z.z+0.125)*(z.z+0.125)
		+ (z.w+0.744)*(z.w+0.744) < 0.0088);
	return main_card & period_2 & smaller_bulb & smaller_bottom & smaller_top;
}

__device__ __forceinline__
void write_to_image(float4 z, float2 temp, int2 ixy, 
	int count, unsigned int *canvas) {
	z.x = z.z;
	z.y = z.w;
	for (int j = 0; j < count; j++) {
		temp.x = z.x*z.x - z.y*z.y + z.z;
		temp.y = 2*z.x*z.y + z.w;
		z.x = temp.x;
		z.y = temp.y;
		write_pixel(temp, ixy, z, canvas);
	}

}

extern "C" {
__global__
void buddha_kernel(unsigned int *canvas, int seed) {
	int idx = blockIdx.x 
		+ threadIdx.x * gridDim.x 
		+ threadIdx.y * gridDim.x * blockDim.x;
	float gridSize = 1/1024.0f;
	int2 ixy;
	float2 temp, coord;
	unsigned int count;
	float4 z;
	float dist;
	hiprandState_t s;
	hiprand_init(seed, idx, 0, &s);

	for (coord.x = 0; coord.x < 1; coord.x += gridSize) {
		for (coord.y = 0; coord.y < 1; coord.y += gridSize) {

			for(int i = 0; i < 10; i++) {

				temp.x = hiprand_uniform(&s);
				temp.y = hiprand_uniform(&s);
				temp *= gridSize;
				temp += coord;

				generate_random_complex(temp, z, dist, count);
				if (check_bulbs(z)) {
					while (count < ITERS & dist < 4) {
						count++;
						temp.x = z.x*z.x - z.y*z.y + z.z;
						temp.y = 2*z.x*z.y + z.w;
						z.x = temp.x;
						z.y = temp.y;
						dist = z.x*z.x + z.y*z.y;
					}

					if (dist > 4) {
						write_to_image(z, temp, ixy, count, canvas);
						z.w *= -1;
						write_to_image(z, temp, ixy, count, canvas);						 
					}
				}
			}
			__syncthreads();
		}
	}
}
}